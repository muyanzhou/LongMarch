﻿#include "function_derivative_test.h"

TEST(Physics, FunctionDerivativeDeterminant3) {
  TestFunctionSet<Determinant3<double>>();
}

TEST(Physics, FunctionDerivativeLogDeterminant3) {
  TestFunctionSet<LogDeterminant3<double>>();
}

TEST(Physics, FunctionDerivativeLogSquareDeterminant3) {
  TestFunctionSet<LogSquareDeterminant3<double>>();
}

TEST(Physics, FunctionDerivativeVecLength) {
  TestFunctionSet<VecLength<double, 3>>();
  TestFunctionSet<VecLength<double, 4>>();
  TestFunctionSet<VecLength<double, 5>>();
}

TEST(Physics, FunctionDerivativeVecNormalized) {
  TestFunctionSet<VecNormalized<double, 3>>();
  TestFunctionSet<VecNormalized<double, 4>>();
  TestFunctionSet<VecNormalized<double, 5>>();
}

TEST(Physics, FunctionDerivativeCross3) {
  TestFunctionSet<Cross3<double>>();
}

TEST(Physics, FunctionDerivativeDot) {
  TestFunctionSet<Dot<double>>();
}

TEST(Physics, FunctionDerivativeCrossNormalized) {
  TestFunctionSet<CrossNormalized<double>>();
}

TEST(Physics, FunctionDerivativeAtan2) {
  TestFunctionSet<Atan2<double>>();
}

TEST(Physics, FunctionDerivativeCrossNorm) {
  TestFunctionSet<CrossNorm<double>>();
}
