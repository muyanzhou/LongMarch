﻿#include "function_derivative_test.h"

TEST(Physics, FunctionDerivativeSphereSDF) {
  SphereSDF<double> f;
  f.center = Eigen::Vector3d::Random();
  f.radius = 3.0;
  TestFunctionSet<SphereSDF<double>>(f);
}

TEST(Physics, FunctionDerivativeLineSDF) {
  for (int i = 0; i < 100; i++) {
    LineSDF<double> f;
    f.A = Eigen::Vector3d::Random();
    f.B = Eigen::Vector3d::Random();
    TestFunctionSet<LineSDF<double>>(f, 1);
  }
}

TEST(Physics, FunctionDerivativeCapsuleSDF) {
  for (int i = 0; i < 100; i++) {
    CapsuleSDF<double> f;
    f.A = Eigen::Vector3d::Random();
    f.B = Eigen::Vector3d::Random();
    f.radius = 3.0;
    TestFunctionSet<CapsuleSDF<double>>(f, 1);
  }
}

TEST(Physics, FunctionDerivativeCubeSDF) {
  for (int i = 0; i < 100; i++) {
    CubeSDF<double> f;
    f.center = {0.0, 0.0, 0.0};
    f.size = 0.1;
    TestFunctionSet<CubeSDF<double>>(f, 1);
  }
}
