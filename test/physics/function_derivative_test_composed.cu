﻿#include "function_derivative_test.h"

TEST(Physics,
     FunctionDerivativeFEMTriangleDeformationGradient3x3RightMulMatrix) {
  Eigen::Matrix2<double> Dm;
  Eigen::Matrix<double, 3, 3> Fp;
  do {
    Dm = Eigen::Matrix2<double>::Random();
  } while (Dm.determinant() < 0);
  Fp = Eigen::Matrix<double, 3, 3>::Random();
  TestFunctionSet<RightMultiplyMatrix<FEMTriangleDeformationGradient3x3<double>,
                                      Eigen::Matrix<double, 3, 3>>>({{Dm}, Fp});
}

TEST(Physics, FunctionDerivativeFEMTriangleNeoHookeanElement) {
  Eigen::Matrix2<double> Dm;
  Eigen::Matrix<double, 3, 3> Fp;
  do {
    Dm = Eigen::Matrix2<double>::Random();
  } while (Dm.determinant() < 0);
  Fp = Eigen::Matrix<double, 3, 3>::Random();
  TestFunctionSet<
      Compose<RightMultiplyMatrix<FEMTriangleDeformationGradient3x3<double>,
                                  Eigen::Matrix<double, 3, 3>>,
              ElasticNeoHookean<double>>>({{{Dm}, Fp}, {1.0, 1.0}});
}
