﻿#include "function_derivative_test.h"

TEST(Physics, FunctionDerivativeDihedralAngleAssistEdgesToNormalsAxis) {
  TestFunctionSet<DihedralAngleAssistEdgesToNormalsAxis<double>>();
}

TEST(Physics, FunctionDerivativeDihedralAngleAssistNormalsAxisToSinCosTheta) {
  TestFunctionSet<DihedralAngleAssistNormalsAxisToSinCosTheta<double>>();
}

TEST(Physics, FunctionDerivativeDihedralAngleByEdges) {
  TestFunctionSet<DihedralAngleByEdges<double>>();
}

TEST(Physics, FunctionDerivativeDihedralAngleAssistVerticesToEdges) {
  TestFunctionSet<DihedralAngleAssistVerticesToEdges<double>>();
}

TEST(Physics, FunctionDerivativeDihedralAngleByVertices) {
  TestFunctionSet<DihedralAngleByVertices<double>>();
}

TEST(Physics, FunctionDerivativeDihedralEnergy) {
  DihedralEnergy<double> f;
  std::random_device rd;
  for (int i = 0; i < 100; i++) {
    f.rest_angle = std::uniform_real_distribution<double>(
        -glm::pi<double>() * 0.5, glm::pi<double>() * 0.5)(rd);
    TestFunctionSet(f, 1);
  }
}

TEST(Physics, FunctionDerivativeDihedralAngle) {
  // Eigen::Vector<double, 12> x;
  // x << 0, 2, 2, 0, 0, 2, 0, 0, -2, -2, 0, -2;
  // TestFunctionSetWithInput<DihedralAngle<double>>({}, x);
  TestFunctionSet<DihedralAngle<double>>({});
}
