﻿#include "function_derivative_test.h"

TEST(Physics, FunctionDerivativeFEMTetrahedronDeformationGradient) {
  Eigen::Matrix3<double> Dm;
  do {
    Dm = Eigen::Matrix3<double>::Random();
  } while (Dm.determinant() < 0);
  TestFunctionSet<FEMTetrahedronDeformationGradient<double>>({Dm});
}

TEST(Physics, FunctionDerivativeFEMDeformationGradient3x2To3x3) {
  TestFunctionSet<FEMDeformationGradient3x2To3x3<double>>();
}

TEST(Physics, FunctionDerivativeFEMTriangleDeformationGradient3x2) {
  Eigen::Matrix2<double> Dm;
  do {
    Dm = Eigen::Matrix2<double>::Random();
  } while (Dm.determinant() < 0);
  TestFunctionSet<FEMTriangleDeformationGradient3x2<double>>({Dm});
}

TEST(Physics, FunctionDerivativeFEMTriangleDeformationGradient3x3) {
  Eigen::Matrix2<double> Dm;
  do {
    Dm = Eigen::Matrix2<double>::Random();
  } while (Dm.determinant() < 0);
  TestFunctionSet<FEMTriangleDeformationGradient3x3<double>>({Dm});
}
