﻿#include "function_derivative_test.h"

TEST(Physics, FunctionDerivativeElasticNeoHookean) {
  TestFunctionSet<ElasticNeoHookean<double>>();
}

TEST(Physics, FunctionDerivativeElasticNeoHookeanSimple) {
  TestFunctionSet<ElasticNeoHookeanSimple<double>>();
}

TEST(Physics, FunctionDerivativeElasticNeoHookeanF3x2) {
  TestFunctionSet<ElasticNeoHookeanF3x2<double>>();
}

TEST(Physics, FunctionDerivativeElasticNeoHookeanSimpleF3x2) {
  TestFunctionSet<ElasticNeoHookeanSimpleF3x2<double>>();
}

TEST(Physics, FunctionDerivativeElasticNeoHookeanTetrahedron) {
  Eigen::Matrix3<double> Dm;
  do {
    Dm = Eigen::Matrix3<double>::Random();
  } while (Dm.determinant() < 0);
  TestFunctionSet<ElasticNeoHookeanTetrahedron<double>>({1.0, 1.0, Dm});
}

TEST(Physics, FunctionDerivativeElasticNeoHookeanSimpleTetrahedron) {
  Eigen::Matrix3<double> Dm;
  do {
    Dm = Eigen::Matrix3<double>::Random();
  } while (Dm.determinant() < 0);
  TestFunctionSet<ElasticNeoHookeanSimpleTetrahedron<double>>({1.0, 1.0, Dm});
}

TEST(Physics, FunctionDerivativeElasticNeoHookeanTriangle) {
  Eigen::Matrix2<double> Dm;
  do {
    Dm = Eigen::Matrix2<double>::Random();
  } while (Dm.determinant() < 0);
  TestFunctionSet<ElasticNeoHookeanTriangle<double>>({1.0, 1.0, Dm});
}

TEST(Physics, FunctionDerivativeElasticNeoHookeanSimpleTriangle) {
  Eigen::Matrix2<double> Dm;
  do {
    Dm = Eigen::Matrix2<double>::Random();
  } while (Dm.determinant() < 0);
  TestFunctionSet<ElasticNeoHookeanSimpleTriangle<double>>({1.0, 1.0, Dm});
}
